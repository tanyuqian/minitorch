#include <hip/hip_runtime.h>

#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32
// typedef float float;

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {

    __shared__ float a_shared[32][32];
    __shared__ float b_shared[32][32];

    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int pi = threadIdx.x;
    int pj = threadIdx.y;

    float accum = 0.0;
    // printf("Start working: %d, %d, %d, %d", i, j, pi, pj);
    for (int k_start = 0; k_start < a_shape[2]; k_start += TILE) {
        int k = k_start + pj;
        if (i < a_shape[1] && k < a_shape[2]) {
            a_shared[pi][pj] = a_storage[a_batch_stride * batch + a_strides[1] * i + a_strides[2] * k];
        } else {
            a_shared[pi][pj] = 0.0;
        }

        k = k_start + pi;
        if (j < b_shape[2] && k < b_shape[1]) {
            b_shared[pi][pj] = b_storage[b_batch_stride * batch + b_strides[1] * k + b_strides[2] * j];
        } else {
            b_shared[pi][pj] = 0.0;
        }

        __syncthreads();

        for (k = 0; k < TILE; ++k) {
            if ((k_start + k) < a_shape[2]) {
                accum += a_shared[pi][k] * b_shared[k][pj];
            }
        }

        __syncthreads();
    }

    if (i < out_shape[1] && j < out_shape[2]) {
        out[out_strides[0] * batch + out_strides[1] * i + out_strides[2] * j] = accum;
    }
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < out_size) {
        to_index(i, out_shape, out_index, shape_size);
        broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
        int o = index_to_position(out_index, out_strides, shape_size);
        int j = index_to_position(in_index, in_strides, shape_size);
        // printf("out[%d] = fn(in[%d]) = %f\n", o, j, fn(fn_id, in_storage[j]));
        out[o] = fn(fn_id, in_storage[j]);
    }
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int out_pos = blockIdx.x * blockDim.x + threadIdx.x;;
    if (out_pos < out_size) {
      out[out_pos] = reduce_value;
      to_index(out_pos, out_shape, out_index, shape_size);
      int o_pos = index_to_position(out_index, out_strides, shape_size);
      // printf("out[%d, %d] = [%f]\n", out_pos, o_pos, out[out_pos]);
      // printf("reduce_dim: %d\n", a_shape[reduce_dim]);
      for(int i = 0; i < a_shape[reduce_dim]; i++) {
        out_index[reduce_dim] = i;
        int i_pos = index_to_position(out_index, a_strides, shape_size);
        out[out_pos] = fn(fn_id, out[out_pos], a_storage[i_pos]);
        // printf("out[%d] = [%f], a_storage[%d] = [%f]\n", out_pos, out[out_pos], i_pos, a_storage[i_pos]);
      }
    }
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < out_size) {
        to_index(i, out_shape, out_index, out_shape_size);
        int o = index_to_position(out_index, out_strides, out_shape_size);
        broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
        int j = index_to_position(a_index, a_strides, a_shape_size);
        broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
        int k = index_to_position(b_index, b_strides, b_shape_size);
        out[o] = fn(fn_id, a_storage[j], b_storage[k]);
    }
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipError_t status = hipMalloc(&d_a, batch * m * n * sizeof(float));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix A Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_b, batch * n * p * sizeof(float));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix B Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_out, batch * m * p * sizeof(float));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix OUT Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    status = hipMalloc(&d_out_shape, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix out_shape Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_out_strides, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix out_strides Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_a_shape, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix a_shape Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_a_strides, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix a_strides Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_b_shape, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix b_shape Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_b_strides, 3 * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Matmul Malloc Matrix b_strides Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }

    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipError_t status = hipMalloc(&d_out, out_size * sizeof(float));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc Matrix out Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_in, in_size * sizeof(float));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc Matrix in Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    status = hipMalloc(&d_out_shape, shape_size * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc out_shape Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_out_strides, shape_size * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc out_strides Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_in_shape, shape_size * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc in_shape Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }
    status = hipMalloc(&d_in_strides, shape_size * sizeof(int));
    if (status != hipSuccess) {
      fprintf(stderr, "Map Malloc in_strides Error: %s\n", hipGetErrorString(status));
      exit(EXIT_FAILURE);
    }

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipError_t status = hipMalloc(&d_a, a_size * sizeof(float));
    status = hipMalloc(&d_b, b_size * sizeof(float));
    status = hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    status = hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    status = hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    status = hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    status = hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    status = hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    status = hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}
