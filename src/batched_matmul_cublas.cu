#include <hipblas.h>
#include <hip/hip_runtime.h>

extern "C"
void batchedMatMulKernel(float **Aarray, float **Barray, float **Carray, int m, int k, int n, int batchCount) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform batched matrix multiplication
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                       m, n, k,
                       &alpha,
                       (const float **)Aarray, m,
                       (const float **)Barray, k,
                       &beta,
                       Carray, m,
                       batchCount);

    hipblasDestroy(handle);
}

