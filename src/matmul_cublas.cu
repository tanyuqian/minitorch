#include <hipblas.h>
#include <hip/hip_runtime.h>

extern "C"
void matmul_cublas(float *a, float *b, float *c, int m, int n, int k) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.;
    float beta = 0.;

    // Note: swapped m and k, and a and b, also using CUBLAS_OP_T for transpose
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, &alpha, b, k, a, m, &beta, c, n);

    hipblasDestroy(handle);
}